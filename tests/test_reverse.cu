/*---------------------------------------------------------------------

Copyright 2019 Paolo G. Crosetto <pacrosetto@gmail.com>
SPDX-License-Identifier: Apache-2.0

---------------------------------------------------------------------*/
#include <boost/program_options.hpp>
#include <cmath>
#include <iostream>
#include <vector>

// boosts timers
#include <boost/timer/timer.hpp>

#include "../src/cupq.h"

// nvcc -gencode=arch=compute_50,code=sm_50 -std=c++14 test_reverse.cu -O3
// -DNDEBUG -lboost_timer -g

int main(int argc, char **argv) {
  using namespace cupq;

  boost::program_options::options_description desc("Allowed options");
  desc.add_options()("help,h", "print usage message")(
      "graph,g", boost::program_options::value<std::string>(),
      "graph filename without extensions");

  boost::program_options::variables_map vm;
  boost::program_options::store(
      boost::program_options::parse_command_line(argc, argv, desc), vm);

  std::string graphname_;

  if (vm.count("help") || !vm.size()) {
    std::cout << "USAGE:\n\n"
                 "dijkstra_simple_exe -g <graph file> \n\n"
                 "printing this help:\n\n"
                 "dijkstra_exe -h \n"
                 "dijkstra_exe \n"
                 "\n";
    return 0;
  }

  if (vm.count("graph")) {
    graphname_ = vm["graph"].as<std::string>();
  } else {
    std::cout << "specify a graph file";
  }

  size_t s;
  hipDeviceGetLimit(&s, hipLimitPrintfFifoSize);
  hipDeviceSetLimit(hipLimitPrintfFifoSize, s * 100);

  using value_t = float;
  using index_t = int;
  using graph_t = Graph<value_t, index_t, cupq::Backend::Host>;

  graph_t graph_;
  graph_.setup();
  graph_.readDIMACS(graphname_.c_str());

  // computing heuristic
  {
    unsigned N = 92;
    std::vector<index_t> sources_(N);
    for (auto i = 0; i < N; ++i) {
      sources_[i] = i + 1;
    }

    std::vector<index_t> destinations_(N);
    for (auto i = 0; i < N; ++i) {
      destinations_[i] = graph_.size() - i - 1;
    }

    auto rgraph_ = graph_.reverse();

    cupq::DijkstraSolver<graph_t> solver_;
    solver_.setSources(sources_);
    solver_.setGraph(graph_);

    solver_.computeDijkstra(-1, -1);
    solver_.finalize();
    auto out = solver_.out();
    graph_.free();
    solver_.setGraph(rgraph_);
    solver_.setSources(destinations_);
    solver_.computeDijkstra(-1, -1);
    auto out3 = solver_.out();
    auto rrgraph_ = rgraph_.reverse();
    rgraph_.free();

    solver_.setGraph(rrgraph_);
    solver_.setSources(sources_);
    solver_.computeDijkstra(-1, -1);
    auto out2 = solver_.out();

    std::cout << "CPU:\n\n";
    for (unsigned i = 0; i < N; ++i) {
      Matrix<value_t, cupq::Backend::Host> d_potential_(graph_.nbNodes() - 1, 1,
                                                        (value_t)FLT_MAX);
      Matrix<index_t, cupq::Backend::Host> d_parent_(graph_.nbNodes() - 1, 1,
                                                     -1);
      std::vector<index_t> orig_{(index_t)(i + 1)};
      dijkstra(graph_, d_potential_, d_parent_, orig_, -1);

      for (unsigned k = 0; k < d_potential_.nrows(); ++k)
        if (out.first(i, k) != d_potential_(k, 0)) {
          if (d_parent_(k, 0) != -1) {
            std::cout << "error66\n";
            std::cout << i << " " << k << " " << out.second(i, k)
                      << " != " << d_parent_(k, 0) << "\n";
            std::cout << out.first(i, k) << " != " << d_potential_(k, 0)
                      << "\n";
          }
        }
    }

    for (auto i = 0; i < N; ++i)
      for (unsigned k = 0; k < graph_.size(); ++k)
        if (out.first(i, k) != out2.first(i, k)) {
          std::cout << "error2\n";
          std::cout << i << " " << k << " " << out.second(i, k)
                    << " != " << out2.second(i, k) << "\n";
          std::cout << out.first(i, k) << " != " << out2.first(i, k) << "\n";
          break;
        }

    auto eps = .1;
    for (auto i = 0; i < N; ++i)
      for (auto j = 0; j < N; ++j)
        if (out.first(i, destinations_[j]) > out3.first(j, sources_[i]) + eps ||
            out.first(i, destinations_[j]) < out3.first(j, sources_[i]) - eps) {
          std::cout << "error3\n";
          std::cout << i << " " << j << " " << out.second(i, destinations_[j])
                    << " != " << out3.second(j, sources_[i]) << "\n";
          std::cout << out.first(i, destinations_[j])
                    << " != " << out3.first(j, sources_[i]) << "\n";
          break;
        }
  }
}
